#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define N 11
#define EDGES 20

void checkCUDAError(const char*);

__global__ void compute_weights(int *d_index_a, int *d_index_b, int *d_weights) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < EDGES) {
        atomicAdd(&d_index_b[i], 1);
    }
}

int main(void) {
    int edge_index[2][20] = {
        {0,1,2,3,0,0,1,4,1,5,1,6,2,7,2,8,3,9,3,10},
        {1,0,0,0,2,3,4,1,5,1,6,1,7,2,8,2,9,3,10,3}
    };
    int* weights;
	int *d_index_a, *d_index_b, *d_nodes, *d_weights;
	unsigned int nodeSize = N * sizeof(int);
	unsigned int edgeSize = EDGES * sizeof(int);

	weights = (int *)malloc(nodeSize);
	hipMalloc((void **)&d_index_a, edgeSize);
	hipMalloc((void **)&d_index_b, edgeSize);
	hipMalloc((void **)&d_nodes, nodeSize);
	hipMalloc((void **)&d_weights, nodeSize);
	checkCUDAError("CUDA malloc");

	hipMemcpy(d_index_a, edge_index[0], edgeSize, hipMemcpyHostToDevice);
	hipMemcpy(d_index_b, edge_index[1], edgeSize, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	compute_weights<<<(EDGES+255)/256, 256>>>(d_index_a, d_index_b, d_weights);
	checkCUDAError("Kernel start");

	hipMemcpy(weights, d_weights, nodeSize, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

    for (int i =0; i< N; ++i){
        printf("%d \n", weights[i]);
    }
	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
