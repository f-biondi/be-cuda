#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define N 2050
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int *a);
void vectorAddCPU(int* a, int* b, int* c, int max);
void validate(int* c, int* c_ref, int max);

__global__ void vectorAdd(int *a, int *b, int *c, int max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

void vectorAddCPU (int *a, int *b, int *c, int max) {
	for(int i=0; i<max; ++i) {
			 c[i] = a[i] + b[i];
	}
}

void validate(int* c, int* c_ref, int max) {
		 int errors = 0;
		 for(int i=0; i<max; ++i) {
					if(c[i] != c_ref[i]) {
							 printf("Error at %d: %d != %d \n", i, c[i], c_ref[i]);
							 errors++;
					}
		 }
		 printf("Total errors: %d \n", errors);
}

int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	unsigned int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	// Launch add() kernel on GPU
    dim3 nBlocks(ceil((float)N / THREADS_PER_BLOCK), 1, 1);
    dim3 nThreads(THREADS_PER_BLOCK, 1, 1);
	vectorAdd <<< nBlocks, nThreads >>>(d_a, d_b, d_c, N);
	checkCUDAError("CUDA kernel");

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

	vectorAddCPU(a, b, c_ref, N);
	validate(c, c_ref, N);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for (unsigned int i = 0; i < N; i++){
		a[i] = rand();
	}
}
