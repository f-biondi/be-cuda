#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define N 11
#define EDGES 20

void checkCUDAError(const char*);
void getMatrix(int* m, int n_nodes);

int main(void) {
    long int edge_index[2][20] = {
        {0,1,2,3,0,0,1,4,1,5,1,6,2,7,2,8,3,9,3,10},
        {1,0,0,0,2,3,4,1,5,1,6,1,7,2,8,2,9,3,10,3}
    };
	long int *d_index_a, *d_index_b, *d_nodes, *d_weights;
	unsigned int nodeSize = N * sizeof(long int);
	unsigned int edgeSize = EDGES * sizeof(long int);

	hipMalloc((void **)&d_index_a, edgeSize);
	hipMalloc((void **)&d_index_b, edgeSize);
	hipMalloc((void **)&d_nodes, nodeSize);
	hipMalloc((void **)&d_weights, nodeSize);
	checkCUDAError("CUDA malloc");

	hipMemcpy(d_index_a, edge_index[0], edgeSize, hipMemcpyHostToDevice);
	hipMemcpy(d_index_b, edge_index[1], edgeSize, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
