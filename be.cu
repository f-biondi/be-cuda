#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define N 11
#define EDGES 20

void checkCUDAError(const char*);

__global__ void compute_weights(int *edge_start, int *edge_end, int *weights, int *node_blocks, int *splitters, int *current_splitter_index) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < EDGES) {
        atomicAdd(
            &weights[edge_end[i]],
            node_blocks[edge_start[i]] == splitters[*current_splitter_index]
        );
    }
}

int main(void) {
    int edge_index[2][20] = {
        {0,1,2,3,0,0,1,4,1,5,1,6,2,7,2,8,3,9,3,10},
        {1,0,0,0,2,3,4,1,5,1,6,1,7,2,8,2,9,3,10,3}
    };
    int *weights, *current_splitter_index;
	int *d_edge_start, *d_edge_end, *d_node_blocks, *d_weights,
        *d_splitters, *d_current_splitter_index, *d_splitters_mask;
	unsigned int nodeSize = N * sizeof(int);
	unsigned int edgeSize = EDGES * sizeof(int);

	weights = (int *)malloc(nodeSize);
	current_splitter_index = (int *)malloc(sizeof(int));
    *current_splitter_index = 0;

	hipMalloc((void **)&d_edge_start, edgeSize);
	hipMalloc((void **)&d_edge_end, edgeSize);
	hipMalloc((void **)&d_node_blocks, nodeSize);
	hipMalloc((void **)&d_weights, nodeSize);
	hipMalloc((void **)&d_splitters, nodeSize);
	hipMalloc((void **)&d_splitters_mask, nodeSize);
	hipMalloc((void **)&d_current_splitter_index, sizeof(int));
	checkCUDAError("CUDA malloc");

	hipMemcpy(d_edge_start, edge_index[0], edgeSize, hipMemcpyHostToDevice);
	hipMemcpy(d_edge_end, edge_index[1], edgeSize, hipMemcpyHostToDevice);
	hipMemcpy(d_current_splitter_index, current_splitter_index, sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");


    while(*current_splitter_index >= 0) {
        hipMemcpy(current_splitter_index, d_current_splitter_index, sizeof(int), hipMemcpyDeviceToHost);
        *current_splitter_index = (*current_splitter_index) - 1;
        hipMemcpy(d_current_splitter_index, current_splitter_index, sizeof(int), hipMemcpyHostToDevice);
        checkCUDAError("CUDA memcpy");

        compute_weights<<<(EDGES+255)/256, 256>>>(d_edge_start, d_edge_end, d_weights, d_node_blocks, d_splitters, d_current_splitter_index);
        checkCUDAError("Compute Weights");

        hipMemcpy(weights, d_weights, nodeSize, hipMemcpyDeviceToHost);
        checkCUDAError("CUDA memcpy");

        for (int i =0; i< N; ++i){
            printf("%d \n", weights[i]);
        }
    }

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
